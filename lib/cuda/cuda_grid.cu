#include "hip/hip_runtime.h"
// Copyright (C) 2011-2012 by the BEM++ Authors
//
// Permission is hereby granted, free of charge, to any person obtaining a copy
// of this software and associated documentation files (the "Software"), to deal
// in the Software without restriction, including without limitation the rights
// to use, copy, modify, merge, publish, distribute, sublicense, and/or sell
// copies of the Software, and to permit persons to whom the Software is
// furnished to do so, subject to the following conditions:
//
// The above copyright notice and this permission notice shall be included in
// all copies or substantial portions of the Software.
//
// THE SOFTWARE IS PROVIDED "AS IS", WITHOUT WARRANTY OF ANY KIND, EXPRESS OR
// IMPLIED, INCLUDING BUT NOT LIMITED TO THE WARRANTIES OF MERCHANTABILITY,
// FITNESS FOR A PARTICULAR PURPOSE AND NONINFRINGEMENT. IN NO EVENT SHALL THE
// AUTHORS OR COPYRIGHT HOLDERS BE LIABLE FOR ANY CLAIM, DAMAGES OR OTHER
// LIABILITY, WHETHER IN AN ACTION OF CONTRACT, TORT OR OTHERWISE, ARISING FROM,
// OUT OF OR IN CONNECTION WITH THE SOFTWARE OR THE USE OR OTHER DEALINGS IN
// THE SOFTWARE.

#include "cuda_grid.hpp"

#include <thrust/gather.h>
#include <thrust/transform.h>
#include <thrust/tabulate.h>
#include <thrust/execution_policy.h>

namespace Bempp {

  struct calculateElementNormalAndIntegrationElementFunctor {

    __host__ __device__
    thrust::tuple<double, double, double, double> operator()(
      const thrust::tuple<double, double, double,
                          double, double, double,
                          double, double, double>& elementCornerCoo) const {

      double vtx0x = thrust::get<0>(elementCornerCoo);
      double vtx0y = thrust::get<1>(elementCornerCoo);
      double vtx0z = thrust::get<2>(elementCornerCoo);

      double vtx1x = thrust::get<3>(elementCornerCoo);
      double vtx1y = thrust::get<4>(elementCornerCoo);
      double vtx1z = thrust::get<5>(elementCornerCoo);

      double vtx2x = thrust::get<6>(elementCornerCoo);
      double vtx2y = thrust::get<7>(elementCornerCoo);
      double vtx2z = thrust::get<8>(elementCornerCoo);

      double nx = (vtx1y - vtx0y) * (vtx2z - vtx0z)
                - (vtx1z - vtx0z) * (vtx2y - vtx0y);

      double ny = (vtx1z - vtx0z) * (vtx2x - vtx0x)
                - (vtx1x - vtx0x) * (vtx2z - vtx0z);

      double nz = (vtx1x - vtx0x) * (vtx2y - vtx0y)
                - (vtx1y - vtx0y) * (vtx2x - vtx0x);

      double integrationElement = std::sqrt(nx*nx + ny*ny + nz*nz);

      nx /= integrationElement;
      ny /= integrationElement;
      nz /= integrationElement;

      return thrust::make_tuple(nx, ny, nz, integrationElement);
    }
  };

  struct funFunctor {

    __host__ __device__
    thrust::tuple<double, double, double> operator()(
      const thrust::tuple<double, double>& localPointCoo) const {

      double r = thrust::get<0>(localPointCoo);
      double s = thrust::get<1>(localPointCoo);

      double fun0 = 1.0 - r - s;
      double fun1 = r;
      double fun2 = s;

      return thrust::make_tuple(fun0, fun1, fun2);
    }
  };

  struct local2globalFunctor {

    unsigned int nLocalPoints;

    thrust::device_ptr<double> vtx0x;
    thrust::device_ptr<double> vtx0y;
    thrust::device_ptr<double> vtx0z;

    thrust::device_ptr<double> vtx1x;
    thrust::device_ptr<double> vtx1y;
    thrust::device_ptr<double> vtx1z;

    thrust::device_ptr<double> vtx2x;
    thrust::device_ptr<double> vtx2y;
    thrust::device_ptr<double> vtx2z;

    thrust::device_ptr<double> fun0;
    thrust::device_ptr<double> fun1;
    thrust::device_ptr<double> fun2;

    local2globalFunctor(
      const unsigned int _nLocalPoints,
      thrust::device_ptr<double> _vtx0x,
      thrust::device_ptr<double> _vtx0y,
      thrust::device_ptr<double> _vtx0z,
      thrust::device_ptr<double> _vtx1x,
      thrust::device_ptr<double> _vtx1y,
      thrust::device_ptr<double> _vtx1z,
      thrust::device_ptr<double> _vtx2x,
      thrust::device_ptr<double> _vtx2y,
      thrust::device_ptr<double> _vtx2z,
      thrust::device_ptr<double> _fun0,
      thrust::device_ptr<double> _fun1,
      thrust::device_ptr<double> _fun2)
      : nLocalPoints(_nLocalPoints),
        vtx0x(_vtx0x), vtx0y(_vtx0y), vtx0z(_vtx0z),
        vtx1x(_vtx1x), vtx1y(_vtx1y), vtx1z(_vtx1z),
        vtx2x(_vtx2x), vtx2y(_vtx2y), vtx2z(_vtx2z),
        fun0(_fun0), fun1(_fun1), fun2(_fun2) {}

    __host__ __device__
    thrust::tuple<double, double, double> operator()(
        const unsigned int i) const {

      // Alternative memory mapping?
      unsigned int localPointIdx = i % nLocalPoints;
      unsigned int elementIdx = i / nLocalPoints;
//      unsigned int localPointIdx = i / nLocalPoints;
//      unsigned int elementIdx = i % nLocalPoints;

      double elVtx0x = vtx0x[elementIdx];
      double elVtx0y = vtx0y[elementIdx];
      double elVtx0z = vtx0z[elementIdx];

      double elVtx1x = vtx1x[elementIdx];
      double elVtx1y = vtx1y[elementIdx];
      double elVtx1z = vtx1z[elementIdx];

      double elVtx2x = vtx2x[elementIdx];
      double elVtx2y = vtx2y[elementIdx];
      double elVtx2z = vtx2z[elementIdx];

      double ptFun0 = fun0[localPointIdx];
      double ptFun1 = fun1[localPointIdx];
      double ptFun2 = fun2[localPointIdx];

      double xGlobal = ptFun0 * elVtx0x
                     + ptFun1 * elVtx1x
                     + ptFun2 * elVtx2x;
      double yGlobal = ptFun0 * elVtx0y
                     + ptFun1 * elVtx1y
                     + ptFun2 * elVtx2y;
      double zGlobal = ptFun0 * elVtx0z
                     + ptFun1 * elVtx1z
                     + ptFun2 * elVtx2z;

      return thrust::make_tuple(xGlobal, yGlobal, zGlobal);
    }
  };

  CudaGrid::CudaGrid() {

    // Initialise member variables
    dim = 0;
    nIdx = 0;
    nVtx = 0;
    nEls = 0;

    d_vertices.clear();
    d_elementCorners.clear();
    d_normals.clear();
    d_integrationElements.clear();
  }

  CudaGrid::~CudaGrid() {

  }

  void CudaGrid::pushGeometry(const Matrix<double> &vertices,
                              const Matrix<int> &elementCorners) {

    // Determine mesh parameters
    dim = vertices.cols();
    nVtx = vertices.rows();
    nIdx = elementCorners.cols();
    nEls = elementCorners.rows();

    std::cout << "nVtx = " << nVtx << std::endl;
    std::cout << "nEls = " << nEls << std::endl;

    if (dim != 3 || nIdx != 3)
      throw std::runtime_error("CudaGrid::pushGeometry(): "
                               "only valid for triangular meshes in three dimensions.");

    // Allocate device memory
    d_vertices.resize(dim * nVtx);
    d_elementCorners.resize(nIdx * nEls);

    // Copy data to device
    d_vertices.assign(vertices.data(), vertices.data()+dim*nVtx);
    d_elementCorners.assign(elementCorners.data(), elementCorners.data()+nIdx*nEls);

//    std::cout << "d_vertices = " << std::endl;
//    for (int i = 0; i < nVtx; ++i) {
//      for (int j = 0; j < dim; ++j) {
//        std::cout << d_vertices[j * nVtx + i] << " " << std::flush;
//      }
//      std::cout << std::endl;
//    }
//    std::cout << std::endl;
//
//    std::cout << "d_elementCorners = " << std::endl;
//    for (int i = 0; i < nEls; ++i) {
//      for (int j = 0; j < nIdx; ++j) {
//        std::cout << d_elementCorners[j * nEls + i] << " " << std::flush;
//      }
//      std::cout << std::endl;
//    }
//    std::cout << std::endl;

    // TEST Calculate element normals vectors and integration elements
    calculateNormalsAndIntegrationElements();

    // TEST Convert local to global coordinates for all elements
    thrust::host_vector<double> localPoints(12);
    localPoints[0] = 0.1;
    localPoints[1] = 0.1;
    localPoints[2] = 0.1;
    localPoints[3] = 0.4;
    localPoints[4] = 0.4;
    localPoints[5] = 0.8;
    localPoints[6] = 0.1;
    localPoints[7] = 0.4;
    localPoints[8] = 0.8;
    localPoints[9] = 0.1;
    localPoints[10] = 0.4;
    localPoints[11] = 0.1;

    thrust::device_vector<double> globalPoints;
    local2global(localPoints, globalPoints);
  }

  void CudaGrid::calculateNormalsAndIntegrationElements() {

    // Allocate device memory
    d_normals.resize(dim * nEls);
    d_integrationElements.resize(nEls);

    // Gather element corner coordinates
    // Perform only once when geometry is pushed to device?
    thrust::device_vector<double> vtx0x(nEls);
    thrust::device_vector<double> vtx0y(nEls);
    thrust::device_vector<double> vtx0z(nEls);

    thrust::device_vector<double> vtx1x(nEls);
    thrust::device_vector<double> vtx1y(nEls);
    thrust::device_vector<double> vtx1z(nEls);

    thrust::device_vector<double> vtx2x(nEls);
    thrust::device_vector<double> vtx2y(nEls);
    thrust::device_vector<double> vtx2z(nEls);

    // Measure time of the GPU execution (CUDA event based)
    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);
    hipEventRecord(start, 0);

    // Vertex 0
    thrust::gather(d_elementCorners.begin(), d_elementCorners.begin()+nEls,
                   d_vertices.begin(),
                   vtx0x.begin());
    thrust::gather(d_elementCorners.begin(), d_elementCorners.begin()+nEls,
                   d_vertices.begin()+nVtx,
                   vtx0y.begin());
    thrust::gather(d_elementCorners.begin(), d_elementCorners.begin()+nEls,
                   d_vertices.begin()+2*nVtx,
                   vtx0z.begin());

    // Vertex 1
    thrust::gather(d_elementCorners.begin()+nEls, d_elementCorners.begin()+2*nEls,
                   d_vertices.begin(),
                   vtx1x.begin());
    thrust::gather(d_elementCorners.begin()+nEls, d_elementCorners.begin()+2*nEls,
                   d_vertices.begin()+nVtx,
                   vtx1y.begin());
    thrust::gather(d_elementCorners.begin()+nEls, d_elementCorners.begin()+2*nEls,
                   d_vertices.begin()+2*nVtx,
                   vtx1z.begin());

    // Vertex 2
    thrust::gather(d_elementCorners.begin()+2*nEls, d_elementCorners.end(),
                   d_vertices.begin(),
                   vtx2x.begin());
    thrust::gather(d_elementCorners.begin()+2*nEls, d_elementCorners.end(),
                   d_vertices.begin()+nVtx,
                   vtx2y.begin());
    thrust::gather(d_elementCorners.begin()+2*nEls, d_elementCorners.end(),
                   d_vertices.begin()+2*nVtx,
                   vtx2z.begin());

    hipEventRecord(stop, 0);
    hipEventSynchronize(stop);
    float elapsedTimeGather;
    hipEventElapsedTime(&elapsedTimeGather , start, stop);
    std::cout << "Time for thrust::gather() in CudaGrid::calculateNormals() is "
      << elapsedTimeGather << " ms" << std::endl;

//    std::cout << "vtx0x = " << std::endl;
//    for (int i = 0; i < nEls; ++i) {
//      std::cout << vtx0x[i] << std::endl;
//    }
//    std::cout << std::endl;

    hipEventRecord(start, 0);

    // Calculate element normals vectors and integration elements
    thrust::transform(
      thrust::make_zip_iterator(
        thrust::make_tuple(vtx0x.begin(), vtx0y.begin(), vtx0z.begin(),
                           vtx1x.begin(), vtx1y.begin(), vtx1z.begin(),
                           vtx2x.begin(), vtx2y.begin(), vtx2z.begin())),
      thrust::make_zip_iterator(
        thrust::make_tuple(vtx0x.end(), vtx0y.end(), vtx0z.end(),
                           vtx1x.end(), vtx1y.end(), vtx1z.end(),
                           vtx2x.end(), vtx2y.end(), vtx2z.end())),
      thrust::make_zip_iterator(
        thrust::make_tuple(d_normals.begin(), d_normals.begin()+nEls,
                           d_normals.begin()+2*nEls, d_integrationElements.begin())),
      calculateElementNormalAndIntegrationElementFunctor());

    hipEventRecord(stop, 0);
    hipEventSynchronize(stop);
    float elapsedTimeNormals;
    hipEventElapsedTime(&elapsedTimeNormals , start, stop);
    std::cout << "Time for calculateNormalsAndIntegrationElements() is "
      << elapsedTimeNormals << " ms" << std::endl;

//    std::cout << "d_normals = " << std::endl;
//    for (int i = 0; i < nEls; ++i) {
//      for (int j = 0; j < dim; ++j) {
//        std::cout << d_normals[j * nEls + i] << " " << std::flush;
//      }
//      std::cout << std::endl;
//    }
//    std::cout << std::endl;
//
//    std::cout << "d_integrationElements = " << std::endl;
//    for (int i = 0; i < nEls; ++i) {
//      std::cout << d_integrationElements[i] << std::endl;
//    }
//    std::cout << std::endl;
  }

  void CudaGrid::local2global(const thrust::host_vector<double> &localPoints,
                              thrust::device_vector<double> &globalPoints) {

    const unsigned int localPointDim = 2;
    const unsigned int nLocalPoints = localPoints.size() / localPointDim;

    std::cout << "nLocalPoints = " << nLocalPoints << std::endl;

    if (localPoints.size() % localPointDim != 0)
      throw std::runtime_error("CudaGrid::local2global(): "
                               "only valid for two-dimensional local points");

    // Allocate device memory
    globalPoints.resize(dim * nEls * nLocalPoints);

    // Evaluate function values on host
    thrust::host_vector<double> h_fun0(nLocalPoints);
    thrust::host_vector<double> h_fun1(nLocalPoints);
    thrust::host_vector<double> h_fun2(nLocalPoints);
    thrust::transform(thrust::host,
        thrust::make_zip_iterator(
            thrust::make_tuple(localPoints.begin(), localPoints.begin()+nLocalPoints)),
        thrust::make_zip_iterator(
            thrust::make_tuple(localPoints.begin()+nLocalPoints, localPoints.end())),
        thrust::make_zip_iterator(
            thrust::make_tuple(h_fun0.begin(), h_fun1.begin(), h_fun2.begin())),
        funFunctor());

    // Copy data to device
    thrust::device_vector<double> fun0 = h_fun0;
    thrust::device_vector<double> fun1 = h_fun1;
    thrust::device_vector<double> fun2 = h_fun2;

//    std::cout << "fun = " << std::endl;
//    for (int i = 0; i < nLocalPoints; ++i) {
//      std::cout << fun0[i] << " "
//                << fun1[i] << " "
//                << fun2[i] << std::endl;
//    }
//    std::cout << std::endl;

    // Gather element corner coordinates
    // Perform only once when geometry is pushed to device?
    thrust::device_vector<double> vtx0x(nEls);
    thrust::device_vector<double> vtx0y(nEls);
    thrust::device_vector<double> vtx0z(nEls);

    thrust::device_vector<double> vtx1x(nEls);
    thrust::device_vector<double> vtx1y(nEls);
    thrust::device_vector<double> vtx1z(nEls);

    thrust::device_vector<double> vtx2x(nEls);
    thrust::device_vector<double> vtx2y(nEls);
    thrust::device_vector<double> vtx2z(nEls);

    // Measure time of the GPU execution (CUDA event based)
    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);
    hipEventRecord(start, 0);

    // Vertex 0
    thrust::gather(d_elementCorners.begin(), d_elementCorners.begin()+nEls,
                   d_vertices.begin(),
                   vtx0x.begin());
    thrust::gather(d_elementCorners.begin(), d_elementCorners.begin()+nEls,
                   d_vertices.begin()+nVtx,
                   vtx0y.begin());
    thrust::gather(d_elementCorners.begin(), d_elementCorners.begin()+nEls,
                   d_vertices.begin()+2*nVtx,
                   vtx0z.begin());

    // Vertex 1
    thrust::gather(d_elementCorners.begin()+nEls, d_elementCorners.begin()+2*nEls,
                   d_vertices.begin(),
                   vtx1x.begin());
    thrust::gather(d_elementCorners.begin()+nEls, d_elementCorners.begin()+2*nEls,
                   d_vertices.begin()+nVtx,
                   vtx1y.begin());
    thrust::gather(d_elementCorners.begin()+nEls, d_elementCorners.begin()+2*nEls,
                   d_vertices.begin()+2*nVtx,
                   vtx1z.begin());

    // Vertex 2
    thrust::gather(d_elementCorners.begin()+2*nEls, d_elementCorners.end(),
                   d_vertices.begin(),
                   vtx2x.begin());
    thrust::gather(d_elementCorners.begin()+2*nEls, d_elementCorners.end(),
                   d_vertices.begin()+nVtx,
                   vtx2y.begin());
    thrust::gather(d_elementCorners.begin()+2*nEls, d_elementCorners.end(),
                   d_vertices.begin()+2*nVtx,
                   vtx2z.begin());

    hipEventRecord(stop, 0);
    hipEventSynchronize(stop);
    float elapsedTimeGather;
    hipEventElapsedTime(&elapsedTimeGather , start, stop);
    std::cout << "Time for thrust::gather() in CudaGrid::local2global() is "
      << elapsedTimeGather << " ms" << std::endl;

    hipEventRecord(start, 0);

    thrust::tabulate(
        thrust::make_zip_iterator(
            thrust::make_tuple(globalPoints.begin(), globalPoints.begin()+nEls*nLocalPoints, globalPoints.begin()+2*nEls*nLocalPoints)),
        thrust::make_zip_iterator(
            thrust::make_tuple(globalPoints.begin()+nEls*nLocalPoints, globalPoints.begin()+2*nEls*nLocalPoints, globalPoints.end())),
        local2globalFunctor(nLocalPoints,
            vtx0x.data(), vtx0y.data(), vtx0z.data(),
            vtx1x.data(), vtx1y.data(), vtx1z.data(),
            vtx2x.data(), vtx2y.data(), vtx2z.data(),
            fun0.data(), fun1.data(), fun2.data()));

    hipEventRecord(stop, 0);
    hipEventSynchronize(stop);
    float elapsedTimeMapping;
    hipEventElapsedTime(&elapsedTimeMapping , start, stop);
    std::cout << "Time for local2global() is "
      << elapsedTimeMapping << " ms" << std::endl;

//    std::cout << "globalPoints = " << std::endl;
//    for (int i = 0; i < nEls; ++i) {
//      for (int j = 0; j < nLocalPoints; ++j) {
//        for (int k = 0; k < dim; ++k) {
//          std::cout << globalPoints[k * nLocalPoints * nEls + i * nLocalPoints + j] << " " << std::flush;
//        }
//        std::cout << std::endl;
//      }
//      std::cout << std::endl;
//    }
//    std::cout << std::endl;
  }

} // namespace Bempp
